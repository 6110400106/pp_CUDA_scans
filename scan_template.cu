
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

#define N 8

__global__ void exclusive_scan(int *d_in) {
	
	__shared__ int temp_in[N];

	int id = threadIdx.x;
        temp_in[id] = d_in[id];
        __syncthreads();

	unsigned int s = 1;
        for(; s <= N-1; s <<= 1) {
                int i = 2 * s * (threadIdx.x + 1) - 1;
		if(i >= s && i < N){
                        //printf("s = %d, i = %d \n", s, i);
                        int a = temp_in[i];
                        int b = temp_in[i-s];
                        __syncthreads();
                        temp_in[i] = a + b;
                }
                __syncthreads();
        }
	
	if(threadIdx.x == 0) {
                temp_in[N-1] = 0;
        }
        for(s = s/2; s >= 1; s >>= 1) {
                int i = 2*s*(threadIdx.x+1)-1;
                if(i >= s && i < N){
                        //printf("s = %d, i = %d \n", s, i);
                        int r = temp_in[i];
                        int l = temp_in[i-s];
                        __syncthreads();
                        temp_in[i] = l + r;
                        temp_in[i-s] = r;
                }
                __syncthreads();
        }
	d_in[id] = temp_in[id];

	//Teacher's code
	/*
	//Phase 1 Uptree
	int s = 1;
	for(; s <= N-1; s <<= 1) {
		int i = 2 * s * (threadIdx.x + 1) - 1;
		if(i-s >= 0 && i < N){
			//printf("s = %d, i = %d \n", s, i);
			int a = d_in[i];
			int b = d_in[i-s];
			__syncthreads();
			d_in[i] = a + b;
		}
		__syncthreads();
	}


	//Phase 2 Downtree
	if(threadIdx.x == 0) {
		d_in[N-1] = 0;
	}
	for(s = s/2; s >= 1; s >>= 1) {
		int i = 2*s*(threadIdx.x+1)-1;
		if(i-s >= 0 && i < N){
                        //printf("s = %d, i = %d \n", s, i);
                        int r = d_in[i];
                        int l = d_in[i-s];
                        __syncthreads();
                        d_in[i] = l + r;
			d_in[i-s] = r;
                }
                __syncthreads();
	}
	*/
}

__global__ void inclusive_scan(int *d_in) {
	
	__shared__ int temp_in[N];

	int i = threadIdx.x;
	temp_in[i] = d_in[i];

	__syncthreads();

	for(unsigned int s = 1; s <= N-1; s <<= 1) {
		if(i >= s && i < N) {
			int a = temp_in[i];
			int b = temp_in[i-s];
			int c = a + b;
			temp_in[i] = c;
		}	
		__syncthreads();
	}

	d_in[i] = temp_in[i];

}

int main()
{
	int h_in[N] = {3, 1, 7, 0, 4, 1, 6, 3};
	int h_out[N];

	//for(int i=0; i < N; i++)
	//	h_in[i] = 1;
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int *d_in;
	//int *d_out;

	hipMalloc((void**) &d_in, N*sizeof(int));
	//cudaMalloc((void**) &d_out, N*sizeof(int));
	hipMemcpy(d_in, &h_in, N*sizeof(int), hipMemcpyHostToDevice);
	
	//Implementing kernel call
	//Timed each kernel call
	hipEventRecord(start);
	//inclusive_scan<<<1, N>>>(d_in);
	exclusive_scan<<<1, N>>>(d_in);
	hipEventRecord(stop);


	hipMemcpy(&h_out, d_in, N*sizeof(int), hipMemcpyDeviceToHost);
	
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	hipFree(d_in);
	//cudaFree(d_out);

	for(int i=0; i<N; i++)
		printf("out[%d] =  %d\n", i, h_out[i]); 
	printf("Time used: %f milliseconds\n", milliseconds);

	return -1;

}
